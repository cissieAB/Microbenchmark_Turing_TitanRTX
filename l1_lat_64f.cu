//This code is a modification of L1 cache benchmark from 
//"Dissecting the NVIDIA Volta GPU Architecture via Microbenchmarking": https://arxiv.org/pdf/1804.06826.pdf

//This benchmark measures the maximum read bandwidth of L1 cache

//This code have been tested on Volta V100 architecture

#include <stdio.h>   
#include <stdlib.h> 
#include <hip/hip_runtime.h>

#define THREADS_NUM 7
#define WARP_SIZE 32
#define L1_SIZE 16384

// GPU error check
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
        if (code != hipSuccess) {
                fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
                if (abort) exit(code);
        }
}

__global__ void l1_bw(uint32_t *startClk, uint32_t *stopClk, double *dsink, double *posArray){
	
	// thread index
	uint32_t tid = threadIdx.x;
	if(tid < THREADS_NUM){
	// a register to avoid compiler optimization
	double sink = 0;
	double *ptr = posArray + tid;
	// populate l1 cache to warm up
	asm volatile ("{\t\n"
		".reg .f64 data;\n\t"
		"ld.global.ca.f64 data, [%1];\n\t"
		"add.f64 %0, data, %0;\n\t"
		"}" : "+d"(sink) : "l"(ptr) : "memory"
	);
	
	// synchronize all threads
	asm volatile ("bar.sync 0;");
	
	// start timing
	uint32_t start = 0;
	asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");
	
	// load data from l1 cache and accumulate
	asm volatile ("{\t\n"
		".reg .f64 data;\n\t"
		"ld.global.ca.f64 data, [%1];\n\t"
		"add.f64 %0, data, %0;\n\t"
		"}" : "+d"(sink) : "l"(ptr) : "memory"
	);

	// synchronize all threads
	asm volatile("bar.sync 0;");
	
	// stop timing
	uint32_t stop = 0;
	asm volatile("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");
	// write time and data back to memory
	startClk[tid] = start;
	stopClk[tid] = stop;
	dsink[tid] = sink;
	}
}

int main(){
	uint32_t *startClk = (uint32_t*) malloc(THREADS_NUM*sizeof(uint32_t));
	uint32_t *stopClk = (uint32_t*) malloc(THREADS_NUM*sizeof(uint32_t));
	double *posArray = (double*) malloc(THREADS_NUM*sizeof(double));
	double *dsink = (double*) malloc(THREADS_NUM*sizeof(double));
	
	uint32_t *startClk_g;
        uint32_t *stopClk_g;
        double *posArray_g;
        double *dsink_g;
	
	for (uint32_t i=0; i<THREADS_NUM; i++)
		posArray[i] = (double)i;
		
	gpuErrchk( hipMalloc(&startClk_g, THREADS_NUM*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&stopClk_g, THREADS_NUM*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&posArray_g, THREADS_NUM*sizeof(double)) );
	gpuErrchk( hipMalloc(&dsink_g, THREADS_NUM*sizeof(double)) );
	
	gpuErrchk( hipMemcpy(posArray_g, posArray, THREADS_NUM*sizeof(double), hipMemcpyHostToDevice) );
	l1_bw<<<1,THREADS_NUM>>>(startClk_g, stopClk_g, dsink_g, posArray_g);
	gpuErrchk( hipMemcpy(startClk, startClk_g, THREADS_NUM*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(stopClk, stopClk_g, THREADS_NUM*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(dsink, dsink_g, THREADS_NUM*sizeof(double), hipMemcpyDeviceToHost) );
	printf("Latency for %d threads = %u \n", THREADS_NUM, stopClk-startClk);

	return 0;
} 
