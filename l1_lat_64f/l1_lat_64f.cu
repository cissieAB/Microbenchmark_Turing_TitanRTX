//This code is a modification of L1 cache benchmark from 
//"Dissecting the NVIDIA Volta GPU Architecture via Microbenchmarking": https://arxiv.org/pdf/1804.06826.pdf

//This benchmark measures the latency of L1 cache 64f reads

//This code have been tested on Volta V100 architecture

#include <stdio.h>   
#include <stdlib.h> 
#include <hip/hip_runtime.h>

#define THREADS_NUM 32
#define WARP_SIZE 32
#define L1_SIZE 32768
#define ITERS 32768

// GPU error check
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
        if (code != hipSuccess) {
                fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
                if (abort) exit(code);
        }
}

//launch 1 block. Measure latency of 32768 reads. 
__global__ void l1_lat(uint32_t *startClk, uint32_t *stopClk, double *posArray){
	
	// thread index
	uint32_t tid = threadIdx.x;
	if(tid < THREADS_NUM){
	// a register to avoid compiler optimization
//	double sink = 0;
	double *ptr = posArray + tid;
	// populate l1 cache to warm up
	asm volatile ("{\t\n"
		".reg .f64 data;\n\t"
		"ld.global.ca.f64 data, [%0];\n\t"
//		"add.f64 %0, data, %0;\n\t"
		"}" : : "l"(ptr) : "memory"
	);
	
	// synchronize all threads
	asm volatile ("bar.sync 0;");
	
	// start timing
	uint32_t start = 0;
	asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");
	for(uint32_t i=0; i<ITERS; ++i) {	
		// load data from l1 cache and accumulate
		asm volatile ("{\t\n"
			".reg .f64 data;\n\t"
			"ld.global.ca.f64 data, [%0];\n\t"
//			"add.f64 %0, data, %0;\n\t"
			"}" :  : "l"(ptr) : "memory"
		);

		// synchronize all threads
		asm volatile("bar.sync 0;");
	}
	// stop timing
	uint32_t stop = 0;
	asm volatile("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");
	// write time and data back to memory
	startClk[tid] = start;
	stopClk[tid] = stop;
	}
}

int main(){
	uint32_t *startClk = (uint32_t*) malloc(THREADS_NUM*sizeof(uint32_t));
	uint32_t *stopClk = (uint32_t*) malloc(THREADS_NUM*sizeof(uint32_t));
	double *posArray = (double*) malloc(THREADS_NUM*sizeof(double));
	double *dsink = (double*) malloc(THREADS_NUM*sizeof(double));
	
	uint32_t *startClk_g;
        uint32_t *stopClk_g;
        double *posArray_g;
        double *dsink_g;
	
	for (uint32_t i=0; i<THREADS_NUM; i++)
		posArray[i] = (double)i;
		
	gpuErrchk( hipMalloc(&startClk_g, THREADS_NUM*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&stopClk_g, THREADS_NUM*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&posArray_g, THREADS_NUM*sizeof(double)) );
	gpuErrchk( hipMalloc(&dsink_g, THREADS_NUM*sizeof(double)) );
	
	gpuErrchk( hipMemcpy(posArray_g, posArray, THREADS_NUM*sizeof(double), hipMemcpyHostToDevice) );

	l1_lat<<<1,THREADS_NUM>>>(startClk_g, stopClk_g, posArray_g);

	gpuErrchk( hipMemcpy(startClk, startClk_g, THREADS_NUM*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(stopClk, stopClk_g, THREADS_NUM*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(dsink, dsink_g, THREADS_NUM*sizeof(double), hipMemcpyDeviceToHost) );
	printf("L1 Latency for %d threads = %u \n", THREADS_NUM, (stopClk[0]-startClk[0]));

	return 0;
} 