#include <stdio.h>   
#include <stdlib.h> 
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 1024
#define THREADS_PER_SM 1024
#define BLOCKS_NUM 1
#define TOTAL_THREADS (THREADS_PER_BLOCK*BLOCKS_NUM)
#define WARP_SIZE 32
#define REPEAT_TIMES 1024

// GPU error check
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
	if (code != hipSuccess) {
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


template <class T>
__global__ void max_flops(uint32_t *startClk, uint32_t *stopClk, T *data1, T *data2, T *res) {
	int gid = blockIdx.x*blockDim.x + threadIdx.x;
	register T s1 = data1[gid];
	register T s2 = data2[gid];
	register T result = 0;

	// synchronize all threads
	asm volatile ("bar.sync 0;");

	// start timing
	uint32_t start = 0;
	asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");

	for (int j=0 ; j<REPEAT_TIMES ; ++j) {
		asm volatile ("{\t\n"
				"fma.rn.f32 %0, %1, %2 , %0;\n\t"
				"}" : "+f"(result),"+f"(s1),"+f"(s2)
		);



	}
	// synchronize all threads
	asm volatile("bar.sync 0;");

	// stop timing
	uint32_t stop = 0;
	asm volatile("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

	// write time and data back to memory
	startClk[gid] = start;
	stopClk[gid] = stop;
	res[gid] = result;
}

int main(){
	uint32_t *startClk = (uint32_t*) malloc(TOTAL_THREADS*sizeof(uint32_t));
	uint32_t *stopClk = (uint32_t*) malloc(TOTAL_THREADS*sizeof(uint32_t));
	float *data1 = (float*) malloc(TOTAL_THREADS*sizeof(float));
	float *data2 = (float*) malloc(TOTAL_THREADS*sizeof(float));
	float *res = (float*) malloc(TOTAL_THREADS*sizeof(float));

	uint32_t *startClk_g;
	uint32_t *stopClk_g;
	float *data1_g;
	float *data2_g;
	float *res_g;

	for (uint32_t i=0; i<TOTAL_THREADS; i++) {
		data1[i] = (float)i;
		data2[i] = (float)i;
	}

	gpuErrchk( hipMalloc(&startClk_g, TOTAL_THREADS*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&stopClk_g, TOTAL_THREADS*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&data1_g, TOTAL_THREADS*sizeof(float)) );
	gpuErrchk( hipMalloc(&data2_g, TOTAL_THREADS*sizeof(float)) );
	gpuErrchk( hipMalloc(&res_g, TOTAL_THREADS*sizeof(float)) );

	gpuErrchk( hipMemcpy(data1_g, data1, TOTAL_THREADS*sizeof(float), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(data2_g, data2, TOTAL_THREADS*sizeof(float), hipMemcpyHostToDevice) );

	max_flops<float><<<BLOCKS_NUM,THREADS_PER_BLOCK>>>(startClk_g, stopClk_g, data1_g, data2_g, res_g);
	gpuErrchk( hipPeekAtLastError() );

	gpuErrchk( hipMemcpy(startClk, startClk_g, TOTAL_THREADS*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(stopClk, stopClk_g, TOTAL_THREADS*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(res, res_g, TOTAL_THREADS*sizeof(float), hipMemcpyDeviceToHost) );

	float flops;
	flops = (float)(REPEAT_TIMES*THREADS_PER_SM*2)/((float)(stopClk[0]-startClk[0]));
	printf("FLOP per SM = %f (flop/clk/SM)\n", flops);
	printf("Total Clk number = %u \n", stopClk[0]-startClk[0]);

	return 0;
} 
