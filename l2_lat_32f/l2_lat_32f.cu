//This code is a modification of L2 cache benchmark from 
//"Dissecting the NVIDIA Volta GPU Architecture via Microbenchmarking": https://arxiv.org/pdf/1804.06826.pdf

//This benchmark measures the read latency of L2 cache

//This code have been tested on Volta V100 architecture

#include <stdio.h>   
#include <stdlib.h> 
#include <hip/hip_runtime.h>

#define TEST_SIZE 1
#define WARP_SIZE 32
#define L1_SIZE 32768
#define THREADS_NUM 1024
#define ARRAY_SIZE (TEST_SIZE+L1_SIZE)

// GPU error check
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
        if (code != hipSuccess) {
                fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
                if (abort) exit(code);
        }
}

__global__ void l1_bw(uint32_t *startClk, uint32_t *stopClk, float *dsink, float *posArray){
	
	// thread index
	uint32_t tid = threadIdx.x;
	if(tid < TEST_SIZE){
	// a register to avoid compiler optimization
	float sink = 0;
	// populate l2 cache to warm up
	if (tid<TEST_SIZE){
		float *ptr = posArray + tid + L1_SIZE;
		asm volatile ("{\t\n"
			".reg .f32 data;\n\t"
			"ld.global.cg.f32 data, [%1];\n\t"
			"add.f32 %0, data, %0;\n\t"
			"}" : "+f"(sink) : "l"(ptr) : "memory"
		);
	}

	// synchronize all threads
	asm volatile ("bar.sync 0;");

	for (uint32_t i=tid; i<L1_SIZE; i+=THREADS_NUM){
                float *ptr = posArray + i;
                asm volatile ("{\t\n"
                        ".reg .f32 data;\n\t"
                        "ld.global.cg.f32 data, [%1];\n\t"
                        "add.f32 %0, data, %0;\n\t"
                        "}" : "+f"(sink) : "l"(ptr) : "memory"
                );
	}
	
        // synchronize all threads
        asm volatile ("bar.sync 0;");

	// start timing
	uint32_t start = 0;
	asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");
	
	// load data from l2 cache and accumulate
	
        if (tid<TEST_SIZE){
                float *ptr = posArray + tid + L1_SIZE;
                asm volatile ("{\t\n"
                        ".reg .f32 data;\n\t"
                        "ld.global.cg.f32 data, [%1];\n\t"
                        "add.f32 %0, data, %0;\n\t"
                        "}" : "+f"(sink) : "l"(ptr) : "memory"
                );
        }

	// synchronize all threads
	asm volatile("bar.sync 0;");
	
	// stop timing
	uint32_t stop = 0;
	asm volatile("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");
	// write time and data back to memory
	startClk[tid] = start;
	stopClk[tid] = stop;
	dsink[tid] = sink;
	}
}

int main(){
	uint32_t *startClk = (uint32_t*) malloc(THREADS_NUM*sizeof(uint32_t));
	uint32_t *stopClk = (uint32_t*) malloc(THREADS_NUM*sizeof(uint32_t));
	float *posArray = (float*) malloc(ARRAY_SIZE*sizeof(float));
	float *dsink = (float*) malloc(THREADS_NUM*sizeof(float));
	
	uint32_t *startClk_g;
        uint32_t *stopClk_g;
        float *posArray_g;
        float *dsink_g;
	
	for (uint32_t i=0; i<ARRAY_SIZE; i++)
		posArray[i] = (float)i;
		
	gpuErrchk( hipMalloc(&startClk_g, THREADS_NUM*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&stopClk_g, THREADS_NUM*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&posArray_g, ARRAY_SIZE*sizeof(float)) );
	gpuErrchk( hipMalloc(&dsink_g, THREADS_NUM*sizeof(float)) );
	
	gpuErrchk( hipMemcpy(posArray_g, posArray, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice) );

	l1_bw<<<1,THREADS_NUM>>>(startClk_g, stopClk_g, dsink_g, posArray_g);

	gpuErrchk( hipMemcpy(startClk, startClk_g, THREADS_NUM*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(stopClk, stopClk_g, THREADS_NUM*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(dsink, dsink_g, THREADS_NUM*sizeof(float), hipMemcpyDeviceToHost) );
	printf("L2 Latency for %d threads = %u \n", TEST_SIZE, stopClk[0]-startClk[0]);

	return 0;
} 